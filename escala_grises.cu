
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#define CHANNELS 3 // canales del rgb (red,green,blue)
using namespace std;


// The input image is encoded as unsigned characters [0, 255]
__global__ void colorConvert(float * Pout, float * Pin, int width, int height)
{ 
  //tenemos almenos tantos threads como numero de pixeles
  int Col = threadIdx.x + blockIdx.x * blockDim.x;
  int Row = threadIdx.y + blockIdx.y * blockDim.y;

  //cada thread trabaja la siguiente seccion de codigo
  //comprobacion de que el thread esta dentro de rango
  if (Col < width && Row < height)
  {
    
    //Linealiza (1D) el array 2d de la imagen 
    //row*width salta la fila completa
    int greyOffset = Row*width + Col;

    //poisicion inicial del color del pixel en el Pin array
    int rgbOffset = greyOffset*CHANNELS;

    //obtencion de los valores rgb separados en 3 arrays
    // g y b son las posiciones siguientes de r almecenadas en memoria global como un vector 1D
    float r = Pin[rgbOffset]; // red value for pixel
    float g = Pin[rgbOffset + 1]; // green value for pixel
    float b = Pin[rgbOffset + 2]; // blue value for pixel

    //Conversion de rgb a escala de grises
    Pout[greyOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
  }
}

//Funcion auxiliar para guardar la imagen .dat
void save_data(float o[225][225])
{
  ofstream archivo("gray.dat");
  for (int i = 0; i < 225; ++i)
  {
    for (int j = 0; j < 225; ++j)
    {
          archivo<<o[i][j]<<" ";
    }
    archivo<<endl;
  }
}

void GrayScale(float m[225][225*3],int width, int height)
{
  float o[225][225];
  //tamaño de la imagen de entrada *3 por el rgb
  int size_in = width * (height*3);
  //tamaño de la imagen de salida
  int size_out = width * height;

  //Calculo para el tamaño de la memoria necesaria del host y device
  int memSize_in = size_in * sizeof(float);
  int memSize_out = size_out * sizeof(float);

  //Direcciones donde se almacenaran los datos del device entrada y salida
  float *d_A, *d_B;

  //Separando memoria para la entrada salida del device
  hipMalloc((void **) &d_A, memSize_in);
  hipMalloc((void **) &d_B, memSize_out);

  //Copia de memoria del host al device (m imagen entrada host, d_A entrada device)
  hipMemcpy(d_A, m, memSize_in, hipMemcpyHostToDevice);

  //Grid 3D (aunque solo se usa 2D) de bloques
  dim3 DimGrid(floor((width-1)/16 + 1), floor((height-1)/16+1), 1);
  //Bloque 3D (aunque solo se usa 2D)de threads
  dim3 DimBlock(16, 16, 1);

  //Llamado al kernel
  colorConvert<<<DimGrid,DimBlock>>>(d_B, d_A, width, height);
 
  //Copia de memoria del device al host (ambos de salida)
  hipMemcpy(o, d_B, memSize_out, hipMemcpyDeviceToHost);

  //Liberacion de memoria del device
  hipFree(d_A);
  hipFree(d_B);
  //Guardando la data en una imagen (.dat)
  save_data(o);
}


//Funcion auxiliar para la lectura de la data
void leer_data(const char *file, float m[225][225*3])
{
  char buffer[100];
  ifstream archivo2("image.dat");
  for (int ii = 0; ii < 225; ++ii)
  {
    for (int jj = 0; jj < 225; ++jj)
    {
          archivo2>>m[ii][jj*3]>>m[ii][jj*3+1]>>m[ii][jj*3+2];
    }
    archivo2.getline(buffer,100);
  }
}


int main()
{

  int width=225, height=225;
  float m[225][225*3];
  leer_data("lena.dat",m);
  GrayScale(m,width,height);
  printf("HECHO\n");
  return EXIT_SUCCESS;

}
