
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <iostream>
#include <float.h>

using namespace std;

#define TILE_WIDTH 32
//#define THREADS_PER_BLOCK 32;

void MatrixMulOnHost(float* M, float* N, float* P, int Width)
{
	for (int i = 0; i < Width; ++i)
		for (int j = 0; j < Width; ++j) {
			float sum = 0;
			for (int k = 0; k < Width; ++k) {
				float a = M[i * Width + k];
				float b = N[k * Width + j];
				sum += a * b;
			}
		P[i * Width + j] = sum;
	}
}

void llenar(int* a, int n)
{
   int i;
   for (i = 0; i < n*n; ++i)
        a[i] = rand()%5+1;
}


__global__ 
void matrixMulti(int *c, int *a, int *b,int n) 
{
    int row = blockIdx.y * blockDim.y + threadIdx.y ; 
    int col = blockIdx.x * blockDim.x + threadIdx.x ;
    if ((row <n) && (col<n))
    {
		int suma=0;
        for(int i=0;i<n;++i)
        {
        	suma+=a[row*n+i]*b[i*n+col];
        }
        c[row*n+col] = suma;	
    } 
}

__global__ void MatrixMulTiled(int * d_P, int * d_M, int* d_N,int Width) 
{
	__shared__ int Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ int Nds[TILE_WIDTH][TILE_WIDTH];
	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	// Identify the row and column of the d_P element to work on
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;
	int Pvalue = 0;
	// Loop over the d_M and d_N tiles required to compute d_P element
	for (int ph = 0; ph < Width/TILE_WIDTH; ++ph) 
	{
		// Collaborative loading of d_M and d_N tiles into shared memory
		if ((Row< Width) && (ph*TILE_WIDTH+tx)< Width)
			Mds[ty][tx] = d_M[Row*Width + ph*TILE_WIDTH + tx];
		if ((ph*TILE_WIDTH+ty)<Width && Col<Width)
			Nds[ty][tx] = d_N[(ph*TILE_WIDTH + ty)*Width + Col];
		__syncthreads();
		for (int k = 0; k < TILE_WIDTH; ++k)
		{
	 		Pvalue += Mds[ty][k] * Nds[k][tx];
		}
	 	__syncthreads();
	}
	d_P[Row*Width + Col] = Pvalue;
}

__global__ void MatrixMulTiledMod(int * d_P, int * d_M, int* d_N,int Width) 
{
	__shared__ int Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ int Nds[TILE_WIDTH][TILE_WIDTH];
	__shared__ int Nds2[TILE_WIDTH][TILE_WIDTH];
	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	// Identify the row and column of the d_P element to work on
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH*2 + tx;
	int Pvalue =0 , Pvalue2=0;
	Mds[ty][tx]=0;
	Nds[ty][tx]=0;
	Nds2[ty][tx]=0;
	__syncthreads(); 

	// Loop over the d_M and d_N tiles required to compute d_P element
	if((Row < Width) && (Col < Width)){
		for (int ph = 0; ph <Width/TILE_WIDTH; ph++) 
		{
			// Collaborative loading of d_M and d_N tiles into shared memory
			//printf("%i - %i -%i \n",ph, Row, Col );
			if ((Row< Width) && (ph*TILE_WIDTH+tx)< Width)
				Mds[ty][tx] = d_M[Row*Width + ph*TILE_WIDTH + tx];
			if ((ph*TILE_WIDTH+ty)<Width && Col<Width)
				Nds[ty][tx] = d_N[(ph*TILE_WIDTH + ty)*Width + Col];
			//printf("%i %i\n",(ph*TILE_WIDTH+ty),Col+TILE_WIDTH);
			if (((ph*TILE_WIDTH + ty)*Width + Col+TILE_WIDTH)<(Width*Width))
			{
				Nds2[ty][tx] = d_N[(ph*TILE_WIDTH + ty)*Width + Col+TILE_WIDTH];
			}
			__syncthreads();
			for (int k = 0; k < TILE_WIDTH; k++)
			{
		 		Pvalue += Mds[ty][k] * Nds[k][tx];
		 		Pvalue2 += Mds[ty][k] * Nds2[k][tx];
			}
		 	__syncthreads();
		}

		d_P[Row*Width + Col] = Pvalue;
		d_P[Row*Width + Col +TILE_WIDTH] = Pvalue2;
	}
}

void printMatrix( int *a , int tam){
	
	for(int i=0;i<tam;i++)
	{
		for(int j=0;j<tam;j++)
		{
			cout<<a[i*tam+j]<<" ";
		}
		cout<<endl;
	}
}

int main(int argc, char *argv[])
{
	srand (time(NULL));
	int  N= strtol(argv[1], NULL, 10);
	int THREADS_PER_BLOCK=TILE_WIDTH;
	//cout<<N<<endl; return 1;
	//printf("Storage size for float : %d \n", sizeof(float));
	//printf("Storage size for int : %d \n", sizeof(int));
	int *a, *b, *c; // host copies of a, b, c
	int *d_a, *d_b, *d_c; //device copies of a,b,c
	//int size = N*N*sizeof(int);
	int size=N*N*sizeof(int);
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = (int *)malloc(size); 
	llenar(a, N);
	
	b = (int *)malloc(size); 
	llenar(b, N);

	c = (int *)malloc(size);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	int blocks= (N + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
	dim3 dimGrid((blocks+THREADS_PER_BLOCK -1)/2, blocks, 1);
	dim3 dimBlock(THREADS_PER_BLOCK,THREADS_PER_BLOCK, 1);
	cout<<"N: "<<N<<"\tBloques : "<<blocks<<"\t Hebras/Bloque: "<<THREADS_PER_BLOCK<<endl; 
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventRecord(start,0);
		//matrixMulti<<<dimGrid,dimBlock>>>(d_c, d_a, d_b, N);
		MatrixMulTiled<<<dimGrid,dimBlock>>>(d_c, d_a, d_b, N);
		//MatrixMulTiledMod<<<dimGrid,dimBlock>>>(d_c, d_a, d_b, N);
		//matrixMulti<<<dimGrid,dimBlock>>>(d_c, d_a, d_b, N);
		//MatrixMulTiled<<<dimGrid,dimBlock>>>(d_c, d_a, d_b, N); cudaEventElapsedTime()
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start,stop);
	hipEventDestroy(start);
    hipEventDestroy(stop);
	printf("Tiempo  : %f ms\n" ,elapsedTime);
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	//cout<<"------A------------"<<endl;
	//printMatrix(a,N);
	//cout<<"------B------------"<<endl;
	//printMatrix(b,N);
	//cout<<"------C------------"<<endl;
	//printMatrix(c,N);
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}