#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <ctime>
#include <time.h>

 

// Kernel CUDA, cada thread trabaja con un elemento de C
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Obtencion del thread id global (en el device)
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    //Comprobacion de no salirse de los limites
    if (id < n)
        c[id] = a[id] + b[id];
}
 

int main( int argc, char* argv[] )
{
    // Tamaño de los vectores
    int n = 100000;
 
    // Vectores de entrada del host
    double *h_a;
    double *h_b;
    // Vector de salida del host
    double *h_c;
 
    // vectores de entrada del device
    double *d_a;
    double *d_b;
    // Vector de salida del device
    double *d_c;
 
    // Tamaño en bytes de cada vector
    size_t bytes = n*sizeof(double);
 
    // Seperando memoria para cada vector del host
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
 
    // Separando memoria para cada vector del device
    //direccion del puntero de la variable, tamaño de memoria a separar
    hipMalloc((void**)&d_a, bytes);
    //cudaMalloc(&d_a, bytes);
    hipMalloc((void**)&d_b, bytes);
    hipMalloc((void**)&d_c, bytes);
 
    int i;
    // Inicializacion de los vectores de entrada del host
    for( i = 0; i < n; i++ ) {
        h_a[i] = sin(i)*sin(4*i);
        h_b[i] = cos(i)*cos(i);
        //printf("%d,%d \n",h_a[i],h_b[i]);
    }
 
    // Copia de los vectores del host al device
    //puntero destino,puntero fuente,numero de bytes a copiar,tipo de copia
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
    //printf("%d,%d \n",d_a[1],d_b[1]);
 
    int blockSize, gridSize;
 
    // numero de threads por bloque
    blockSize = 1024;
 
    // Numero de bloques en el grid
    gridSize = (int)ceil((float)n/blockSize);
 
    //Toma de tiempo
    //unsigned double timer = 0.0;
    //cutCreateTimer (& timer ) ;
	//cutStartTimer ( timer ) ;


    // Ejecucion del kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
 	

    //cudaThreadSynchronize () ;
	//cutStopTimer ( timer ) ;
	//printf (" CUDA execution time = %f ms\n", cutGetTimerValue ( timer ) ) ;

    // Copia del resultado al host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost);
 	//printf("HOLA%d \n",h_c[100000-1]);


    // Suma del vector y promedio del mismo
    double sum = 0;
    for(i=0; i<n; i++){
    	//printf("%d \n",h_c[i]);
        sum += h_c[i];

    }
    
    printf("final result: %f\n", sum/(double)n);
 
    // Liberando memoria del device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    // Liberando memoria del Host
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}
