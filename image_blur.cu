
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
#include <iostream>
#define BLUR_SIZE 3
using namespace std;

__global__
void blurKernel(float * in, float * out, int w, int h)
{
  //Obtencion de los datos del thread en X y Y
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  int Row = blockIdx.y * blockDim.y + threadIdx.y;

  //Comprobando que el thread este dentro de los limites
  if (Col < w && Row < h)
  {
    int pixVal = 0;
    int pixels = 0;

    //Calculando el promedio del valor de la sub matriz del pixel de 2xBLUR_SIZE x 2xBLUR_SIZE
    //EJE Y
    for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow)
    {
      //EJE X
      for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol)
      {
        //Calculo de la posicion actual
        int curRow = Row + blurRow;
        int curCol = Col + blurCol;

        //Comprobacion de los limites
        if(curRow > -1 && curRow < h && curCol > -1 && curCol < w)
        {
          //Linealizando la sub matriz de 2xBLUR_SIZE x 2xBLUR_SIZE
          //Estos valores seran usados para el promedio 
          pixVal += in[curRow * w + curCol];
          pixels++; // Contador del numero de pixeles usados para el blur
        }
      }
    }
    // Write our new pixel value out
    // Escribiendo el nuevo valor del pixel segun el promedio de los datos calculados anteriormente
    //Acceso lineal a la memoria para la salida
    out[Row * w + Col] = (float)(pixVal / pixels);
  }
}


//Funcion auxiliar para guardar la data en una imagen .dat
void save_data(float r[225][225], float g[225][225], float b[225][225])
{
  ofstream archivo("bluur.dat");
  for (int i = 0; i < 225; ++i)
  {
    for (int j = 0; j < 225; ++j)
    {
          archivo<<r[i][j]<<" "<<g[i][j]<<" "<<b[i][j]<<" ";
    }
    archivo<<endl;
  }
}


void Blur(float r[225][225], float g[225][225], float b[225][225], int width, int height)
{
  //Vectores de salida rgb
  float o_r[225][225];
  float o_g[225][225];
  float o_b[225][225];

  //tamaño de imagen
  int size = width * height;

  //cantidad de memoria necesaria
  int memSize = size * sizeof(float);

  //Direcciones de memoria del device
  float *d_A, *d_B;

  //Separando memoria en el device
  hipMalloc((void **) &d_A, memSize);
  hipMalloc((void **) &d_B, memSize);

  //COLOR ROJO
  //Copia del host al device
  hipMemcpy(d_A, r, memSize, hipMemcpyHostToDevice);

  //Grid 3D (aunque solo se usa 2D) de bloques
  dim3 DimGrid(floor((width-1)/16 + 1), floor((height-1)/16+1), 1);
  //Bloque 3D de threads
  dim3 DimBlock(16, 16, 1);
  //Llamado al kernel
  blurKernel<<<DimGrid,DimBlock>>>(d_A, d_B, width, height);
  //Copia de memoria del device al host (Salida Roja)
  hipMemcpy(o_r, d_B, memSize, hipMemcpyDeviceToHost);


  //COLOR VERDE
  hipMemcpy(d_A, g, memSize, hipMemcpyHostToDevice);

  //Llamado al kernel
  blurKernel<<<DimGrid,DimBlock>>>(d_A, d_B, width, height);
  //Copia de memoria del device al host (Salida Verde)
  hipMemcpy(o_g, d_B, memSize, hipMemcpyDeviceToHost);

  //Copia del host al device
  hipMemcpy(d_A, b, memSize, hipMemcpyHostToDevice);



  //COLOR AZUL
  //Llamado al kernel 
  blurKernel<<<DimGrid,DimBlock>>>(d_A, d_B, width, height);

  //Copia del device al host (salida Azul)
  hipMemcpy(o_b, d_B, memSize, hipMemcpyDeviceToHost);

  //Liberando memoria del device
  hipFree(d_A);
  hipFree(d_B);
  //Guardar la data en imagen .dat
  save_data(o_r,o_g,o_b);
}


//Funcion de apoyo para la lectura de la imagen
void leer_data(const char *file, float r[225][225], float g[225][225], float b[225][225])
{
  char buffer[100];
  ifstream archivo2("lena.dat");
  for (int ii = 0; ii < 225; ++ii)
  {
    for (int jj = 0; jj < 225; ++jj)
    {
          archivo2>>r[ii][jj]>>g[ii][jj]>>b[ii][jj];
    }
    archivo2.getline(buffer,100);
  }
}


int main()
{
  int width=225, height=225;
  float r[225][225];
  float g[225][225];
  float b[225][225];
  leer_data("lena.dat",r,g,b);
  Blur(r,g,b,width,height);
  printf("HECHO\n");
  return EXIT_SUCCESS;
}
